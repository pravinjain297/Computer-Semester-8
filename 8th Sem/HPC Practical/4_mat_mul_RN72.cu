/*
Title:-Write a CUDA Program for :
1. Matrix Multiplication using CUDA C
Name:-Pravin Jain
Roll NO:-72
Subject:-LP-V
*/
#include<hip/hip_runtime.h>
#include<stdio.h>

int main(void)
 {
    void MatrixMultiplication(float *, float *, float *, int);

    //const int Width = 5;

    float M[5*5], N[5*5], P[5*5];

    for(int i = 0; i < (5*5) ; i++)
    {
        M[i] = 5;
        N[i] = 5;
        P[i] = 0;
    }
    
 MatrixMultiplication(M, N, P, 5);

    for(int i = 0; i < (5*5) ; i++)
    {
        printf("%f \n", P[i]);
    }

    return 0;
}

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int width)
 {
    //2D Thread ID
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

    for(int k = 0; k < 5 ; k++)
    {
        float Mdelement = Md[ty*5 + k];
        float Ndelement = Nd[k*5 + tx];
        Pvalue = Pvalue + (Mdelement*Ndelement);
    }

    Pd[ty*5 + tx] = Pvalue;
 }

void MatrixMultiplication(float *M, float *N, float *P, int width)
  {
    int size = 5*5*sizeof(float);

    float *Md, *Nd, *Pd;

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);

    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock(5,5);
    dim3 dimGrid(1,1);

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,5);

    //Transfer P from device to host
    hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
 }



/*
OUTPUT:

sl-sys11@slsys11-OptiPlex-3010:~$ nvcc matp.cu
sl-sys11@slsys11-OptiPlex-3010:~$ ./a.out
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
125.000000 
sl-sys11@slsys11-OptiPlex-3010:~$ 

*/

